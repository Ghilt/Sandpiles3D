#include "hip/hip_runtime.h"
/*
* This code is taken more or less entirely from the NVIDIA CUDA SDK.
* This software contains source code provided by NVIDIA Corporation.
*
*/
//https://github.com/kunzmi/managedCuda/wiki/Setup-a-managedCuda-project

//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>


extern "C" {
	// Device code
	/*IDE will complain about __constant__ and other cuda keywords*/
	__constant__ int maxVal;
	__constant__ int n;
	__constant__ int n2;
	__constant__ int n3;

	__device__ void gainGrainPositive(int changedDimension, int coordNeighbour, int coord, const int* origin, int* delta)
	{
		if (changedDimension < n) {
			if (coordNeighbour < n3 && origin[coordNeighbour] >= maxVal) {
				delta[coord]++;
			}
		}
	}

	__device__ void gainGrainNegative(int changedDimension, int coordNeighbour, int coord, const int* origin, int* delta)
	{
		if (changedDimension >= 0) {
			if (coordNeighbour >= 0 && origin[coordNeighbour] >= maxVal) {
				delta[coord]++;
			}
		}
	}

	__global__ void CalculateSandpilesDelta(const int* origin, int* delta, int* nextIteration)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		int y = blockDim.y * blockIdx.y + threadIdx.y;
		int z = blockDim.z * blockIdx.z + threadIdx.z;
		int coord = x * n * n + y * n + z;

		if (origin[coord] >= maxVal)
		{
			delta[coord] -= maxVal;
		}
		/*Bug which should be tried out visually: Let coordinates individual x,y,z component be negative and continue with flow */

		int xN = x - 1;
		int xP = x + 1;
		int yN = y - 1;
		int yP = y + 1;
		int zN = z - 1;
		int zP = z + 1;

		int coordL = xN * n2 + y * n + z;
		int coordR = xP * n2 + y * n + z; //these calculated unecessarily if inbounds is false, possible optimization
		int coordD = x * n2 + yN * n + z;
		int coordU = x * n2 + yP * n + z;
		int coordB = x * n2 + y * n + zN;
		int coordF = x * n2 + y * n + zP;

		gainGrainNegative(xN, coordL, coord, origin, delta);
		gainGrainPositive(xP, coordR, coord, origin, delta);
		gainGrainNegative(yN, coordD, coord, origin, delta);
		gainGrainPositive(yP, coordU, coord, origin, delta);
		gainGrainNegative(zN, coordB, coord, origin, delta);
		gainGrainPositive(zP, coordF, coord, origin, delta);

		nextIteration[coord] = origin[coord] + delta[coord];


		//if (xP >= 0) { // this more effiecient i think, but scrapped for readability
		//  int coordR = xP * n2 + y * n + z;
		//	if (coordR >= 0 && origin[coordR] >= maxVal) {
		//		delta[coord]++;
		//	}
		//}
		/*if (xN < n) {
		if (coordL < n3 && origin[coordL] >= maxVal) {
		delta[coord]++;
		}
		}

		if (coordU < n3 && origin[coordU] >= maxVal) {
		delta[coord]++;
		}
		if (coordD >= 0 && origin[coordD] >= maxVal) {
		delta[coord]++;
		}
		if (coordB >= 0 && origin[coordB] >= maxVal) {
		delta[coord]++;
		}
		if (coordF < n3 && origin[coordF] >= maxVal) {
		delta[coord]++;
		}*/

	}

	__global__ void CalculateSandpilesDeltaThreadPerZColumn(const int* origin, int* delta, int* nextIteration)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		int y = blockDim.y * blockIdx.y + threadIdx.y;
		int z = 0;
		while (z < n) {
			int coord = x * n * n + y * n + z;

			if (origin[coord] >= maxVal)
			{
				delta[coord] -= maxVal;
			}
			int xN = x - 1;
			int xP = x + 1;
			int yN = y - 1;
			int yP = y + 1;
			int zN = z - 1;
			int zP = z + 1;

			int coordL = xN * n2 + y * n + z;
			int coordR = xP * n2 + y * n + z; //these calculated unecessarily if inbounds is false, possible optimization
			int coordD = x * n2 + yN * n + z;
			int coordU = x * n2 + yP * n + z;
			int coordB = x * n2 + y * n + zN;
			int coordF = x * n2 + y * n + zP;

			gainGrainNegative(xN, coordL, coord, origin, delta);
			gainGrainPositive(xP, coordR, coord, origin, delta);
			gainGrainNegative(yN, coordD, coord, origin, delta);
			gainGrainPositive(yP, coordU, coord, origin, delta);
			gainGrainNegative(zN, coordB, coord, origin, delta);
			gainGrainPositive(zP, coordF, coord, origin, delta);

			nextIteration[coord] = origin[coord] + delta[coord];

			z++;
		}
	}

	__global__ void CalculateSandpilesDeltaThreadPerZColumnOptimized(const int* origin, int* delta, int* nextIteration)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		int y = blockDim.y * blockIdx.y + threadIdx.y;
		int z = 0;
		while (z < n) {
			int coord = x * n2 + y * n + z;

			if (origin[coord] >= maxVal)
			{
				delta[coord] -= maxVal;
			}
			int xN = x - 1;
			int xP = x + 1;
			int yN = y - 1;
			int yP = y + 1;
			int zN = z - 1;
			int zP = z + 1;

			if (xN >= 0) { // possible optimization as the X term is the biggest we do not need to check it 
				int coordL = xN * n2 + y * n + z;
				if (coordL >= 0 && origin[coordL] >= maxVal) {
					delta[coord]++;
				}
			}
			if (xP < n) {
				int coordR = xP * n2 + y * n + z;
				if (coordR < n3 && origin[coordR] >= maxVal) {
					delta[coord]++;
				}
			}
			if (yN >= 0) {
				int coordD = x * n2 + yN * n + z;
				if (coordD >= 0 && origin[coordD] >= maxVal) {
					delta[coord]++;
				}
			}
			if (yP < n) {
				int coordU = x * n2 + yP * n + z;
				if (coordU < n3 && origin[coordU] >= maxVal) {
					delta[coord]++;
				}
			}
			if (zN >= 0) {
				int coordB = x * n2 + y * n + zN;
				if (coordB >= 0 && origin[coordB] >= maxVal) {
					delta[coord]++;
				}
			}
			if (zP < n) {
				int coordF = x * n2 + y * n + zP;
				if (coordF < n3 && origin[coordF] >= maxVal) {
					delta[coord]++;
				}
			}
			nextIteration[coord] = origin[coord] + delta[coord];

			z++;
		}
	}

}