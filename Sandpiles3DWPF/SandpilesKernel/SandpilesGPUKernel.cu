#include "hip/hip_runtime.h"
/*
* This code is taken more or less entirely from the NVIDIA CUDA SDK.
* This software contains source code provided by NVIDIA Corporation.
*
*/
//https://github.com/kunzmi/managedCuda/wiki/Setup-a-managedCuda-project

//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>


extern "C" {
	// Device code
	/*IDE will complain about __constant__ and other cuda keywords*/
	__constant__ int maxVal;
	__constant__ int side;
	__constant__ int depth;
	__constant__ int sideTimesDepth;
	__constant__ int size;

	__global__ void CalculateSandpilesDeltaThreadPerZColumnOptimized(const int* origin, int* delta, int* nextIteration)
	{
		int x = blockDim.x * blockIdx.x + threadIdx.x;
		int y = blockDim.y * blockIdx.y + threadIdx.y;
		int coordPart = x * sideTimesDepth + y * depth;
		int xN = x - 1;
		int xP = x + 1;
		int yN = y - 1;
		int yP = y + 1;
		bool xNInBounds = xN >= 0;
		bool xPInBounds = xP < side;
		bool yNInBounds = yN >= 0;
		bool yPInBounds = yP < side;
		int xNCoordPart = xN * sideTimesDepth + y * depth;
		int xPCoordPart = xP * sideTimesDepth + y * depth;
		int yNCoordPart = x * sideTimesDepth + yN * depth;
		int yPCoordPart = x * sideTimesDepth + yP * depth;
		int z = 0;
		while (z < depth) {
			int coord = coordPart + z;
			int zN = z - 1;
			int zP = z + 1;

			if (origin[coord] >= maxVal)
			{
				delta[coord] -= maxVal;
			}

			if (xNInBounds) { // possible optimization as the X term is the biggest we do not need to check it 
				int coordL = xNCoordPart + z;
				if (coordL >= 0 && origin[coordL] >= maxVal) {
					delta[coord]++;
				}
			}
			if (xPInBounds) {
				int coordR = xPCoordPart + z;
				if (coordR < size && origin[coordR] >= maxVal) {
					delta[coord]++;
				}
			}
			if (yNInBounds) {
				int coordD = yNCoordPart + z;
				if (coordD >= 0 && origin[coordD] >= maxVal) {
					delta[coord]++;
				}
			}
			if (yPInBounds) {
				int coordU = yPCoordPart + z;
				if (coordU < size && origin[coordU] >= maxVal) {
					delta[coord]++;
				}
			}
			if (zN >= 0) {
				int coordB = coordPart + zN;
				if (coordB >= 0 && origin[coordB] >= maxVal) {
					delta[coord]++;
				}
			}
			if (zP < depth) {
				int coordF = coordPart + zP;
				if (coordF < size && origin[coordF] >= maxVal) {
					delta[coord]++;
				}
			}
			nextIteration[coord] = origin[coord] + delta[coord];

			z++;
		}
	}

	/*
	Comment block below was other prettier versions for experimenting with speeds/optimization
	They have NOT been updated to accomodate a different depth for the z parameter
	*/
	//__device__ void gainGrainPositive(int changedDimension, int coordNeighbour, int coord, const int* origin, int* delta)
	//{
	//	if (changedDimension < side) {
	//		if (coordNeighbour < size && origin[coordNeighbour] >= maxVal) {
	//			delta[coord]++;
	//		}
	//	}
	//}

	//__device__ void gainGrainNegative(int changedDimension, int coordNeighbour, int coord, const int* origin, int* delta)
	//{
	//	if (changedDimension >= 0) {
	//		if (coordNeighbour >= 0 && origin[coordNeighbour] >= maxVal) {
	//			delta[coord]++;
	//		}
	//	}
	//}

	//__global__ void CalculateSandpilesDelta(const int* origin, int* delta, int* nextIteration)
	//{
	//	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//	int y = blockDim.y * blockIdx.y + threadIdx.y;
	//	int z = blockDim.z * blockIdx.z + threadIdx.z;
	//	int coord = x * side * side + y * side + z;

	//	if (origin[coord] >= maxVal)
	//	{
	//		delta[coord] -= maxVal;
	//	}
	//	/*Bug which should be tried out visually: Let coordinates individual x,y,z component be negative and continue with flow */

	//	int xN = x - 1;
	//	int xP = x + 1;
	//	int yN = y - 1;
	//	int yP = y + 1;
	//	int zN = z - 1;
	//	int zP = z + 1;

	//	int coordL = xN * sideTimesDepth + y * side + z;
	//	int coordR = xP * sideTimesDepth + y * side + z; //these calculated unecessarily if inbounds is false, possible optimization
	//	int coordD = x * sideTimesDepth + yN * side + z;
	//	int coordU = x * sideTimesDepth + yP * side + z;
	//	int coordB = x * sideTimesDepth + y * side + zN;
	//	int coordF = x * sideTimesDepth + y * side + zP;

	//	gainGrainNegative(xN, coordL, coord, origin, delta);
	//	gainGrainPositive(xP, coordR, coord, origin, delta);
	//	gainGrainNegative(yN, coordD, coord, origin, delta);
	//	gainGrainPositive(yP, coordU, coord, origin, delta);
	//	gainGrainNegative(zN, coordB, coord, origin, delta);
	//	gainGrainPositive(zP, coordF, coord, origin, delta);

	//	nextIteration[coord] = origin[coord] + delta[coord];


	//	//if (xP >= 0) { // this more effiecient i think, but scrapped for readability
	//	//  int coordR = xP * n2 + y * n + z;
	//	//	if (coordR >= 0 && origin[coordR] >= maxVal) {
	//	//		delta[coord]++;
	//	//	}
	//	//}
	//	/*if (xN < n) {
	//	if (coordL < n3 && origin[coordL] >= maxVal) {
	//	delta[coord]++;
	//	}
	//	}

	//	if (coordU < n3 && origin[coordU] >= maxVal) {
	//	delta[coord]++;
	//	}
	//	if (coordD >= 0 && origin[coordD] >= maxVal) {
	//	delta[coord]++;
	//	}
	//	if (coordB >= 0 && origin[coordB] >= maxVal) {
	//	delta[coord]++;
	//	}
	//	if (coordF < n3 && origin[coordF] >= maxVal) {
	//	delta[coord]++;
	//	}*/

	//}

	//__global__ void CalculateSandpilesDeltaThreadPerZColumn(const int* origin, int* delta, int* nextIteration)
	//{
	//	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//	int y = blockDim.y * blockIdx.y + threadIdx.y;
	//	int z = 0;
	//	while (z < side) {
	//		int coord = x * side * side + y * side + z;

	//		if (origin[coord] >= maxVal)
	//		{
	//			delta[coord] -= maxVal;
	//		}
	//		int xN = x - 1;
	//		int xP = x + 1;
	//		int yN = y - 1;
	//		int yP = y + 1;
	//		int zN = z - 1;
	//		int zP = z + 1;

	//		int coordL = xN * sideTimesDepth + y * side + z;
	//		int coordR = xP * sideTimesDepth + y * side + z; //these calculated unecessarily if inbounds is false, possible optimization
	//		int coordD = x * sideTimesDepth + yN * side + z;
	//		int coordU = x * sideTimesDepth + yP * side + z;
	//		int coordB = x * sideTimesDepth + y * side + zN;
	//		int coordF = x * sideTimesDepth + y * side + zP;

	//		gainGrainNegative(xN, coordL, coord, origin, delta);
	//		gainGrainPositive(xP, coordR, coord, origin, delta);
	//		gainGrainNegative(yN, coordD, coord, origin, delta);
	//		gainGrainPositive(yP, coordU, coord, origin, delta);
	//		gainGrainNegative(zN, coordB, coord, origin, delta);
	//		gainGrainPositive(zP, coordF, coord, origin, delta);

	//		nextIteration[coord] = origin[coord] + delta[coord];

	//		z++;
	//	}
	//}

}